#include "CudaException.h"

#include <cassert>
#include <cstddef>
#include <exception>
#include <sstream>
#include <string>

#include <hip/hip_runtime.h>

namespace caracal {
CudaException::CudaException(hipError_t error, std::string &&message)
    : error(error), message(move(message)) {
  assert(error != hipSuccess);
}

CudaException::~CudaException() {}

hipError_t CudaException::GetError() const noexcept { return error; }

const char *CudaException::what() const noexcept { return message.c_str(); }

std::string CudaException::BuildMessage(const char *filename,
                                        size_t line_number,
                                        const char *function_name,
                                        hipError_t error) {
  std::stringstream stream;
  stream << "CUDA error at " << filename << ':' << line_number << " ("
         << function_name << "): " << hipGetErrorName(error) << ": "
         << hipGetErrorString(error);
  return stream.str();
}
} // namespace caracal