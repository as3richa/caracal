#include "hip/hip_runtime.h"
#include "GenerateRandomPlanes.h"

#include <cstdint>

#include <hiprand/hiprand_kernel.h>

namespace caracal {

__global__ static void GenerateRandomPlanesKernel(float *planes, size_t count,
                                                  size_t dimensions,
                                                  size_t pitch, uint64_t seed) {
  const size_t x = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t y = threadIdx.y + blockIdx.y * blockDim.y;
  if (y >= count || x >= dimensions) {
    return;
  }

  const size_t i = x + y * (pitch / sizeof(float));
  hiprandState_t state;
  hiprand_init(seed, i, 0, &state);
  planes[i] = hiprand_uniform(&state) - 0.5;
}

hipError_t GenerateRandomPlanes(float **planes, size_t *pitch, size_t count,
                                 size_t dimensions, uint64_t seed) {
  hipError_t error;

  error = hipMallocPitch((void **)planes, pitch, dimensions * sizeof(float),
                          count);
  if (error != hipSuccess) {
    return error;
  }

  const dim3 block(16, 16, 1);
  const dim3 grid((dimensions + block.x - 1) / block.x,
                  (count + block.y - 1) / block.y, 1);

  GenerateRandomPlanesKernel<<<grid, block>>>(*planes, count, dimensions,
                                              *pitch, seed);
  error = hipGetLastError();
  if (error != hipSuccess) {
    hipFree(*planes);
    return error;
  }

  return hipSuccess;
}

} // namespace caracal